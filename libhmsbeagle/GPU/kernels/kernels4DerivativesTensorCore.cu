#include "hip/hip_runtime.h"

KW_GLOBAL_KERNEL void kernelPartialsPartialsGrowing(KW_GLOBAL_VAR REAL* KW_RESTRICT partials1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials2,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT partials3,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices1,
                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT matrices2,
//                                                    KW_GLOBAL_VAR REAL* KW_RESTRICT tmpAcc,
                                                    int endPattern) {
#ifdef FW_OPENCL_CPU // CPU/MIC implementation
    todo(); // TODO
#else // GPU implementation
    DETERMINE_INDICES_4_GPU();

    const int WMMA_M = 8;
    const int WMMA_N = 8;
    const int WMMA_K = 4;

    int y = deltaPartialsByState + deltaPartialsByMatrix;
    KW_LOCAL_MEM REAL sPartials1[PATTERN_BLOCK_SIZE * 4 * 4];
//    KW_LOCAL_MEM REAL sPartials1Tmp[PATTERN_BLOCK_SIZE * 4 * 4];
    KW_LOCAL_MEM REAL sPartials2[PATTERN_BLOCK_SIZE * 4 * 4];

    const int warpSize = 32;
    const int permuteXSize = 32;
    const int totalWarps = (PATTERN_BLOCK_SIZE * 4 * 4)/permuteXSize;
    const int contiguousX = 4;
    const int totalBankGroups = permuteXSize/contiguousX;

    int warpState = tx / warpSize;
    int warpPattern = patIdx;
    int warpIdx = warpState + warpPattern * 0.5; // blockDim.x is half a warp
    int laneId = tx + (patIdx % 2) * 16;

    int ty = KW_LOCAL_ID_1;

// Indices to permute ShM for partials
// X -> threadIdx.x or state and Y -> threadIdx.y or patIdx
// (int(X/8): Splits 32 values into groups of 4.
// ((Y & 1) * -2 + 1)): For strip-mined layout: If patIdx is even increment by 1 else by -1
// & 0x07 To cycle within the limits [0,1,2,3,4,5,6,7] i.e., [0, ... , PADDED_STATE_COUNT/WMMA_K]
#define GET_SMEM_ROW_PARTIALS(X, Y) ( (X / contiguousX) + ((Y/4) * 4))
#define GET_BANK_GROUP_PARTIALS(X,Y) ((Y + (X/contiguousX) * (0 - (Y & 1) | 1)) & (4 - 1) )
#define GET_SMEM_COL_PARTIALS(X,Y) (GET_BANK_GROUP_PARTIALS(X,Y) * contiguousX + (X % contiguousX))
#define GET_SMEM_OFFSET_PARTIALS(X,Y) (GET_SMEM_ROW_PARTIALS(X, Y) * 16 + GET_SMEM_COL_PARTIALS(X, Y))

    /* copy PADDED_STATE_COUNT * PATTERN_BLOCK_SIZE lengthed partials*/
    if (pattern < endPattern) {
        // Read in permuted for partials1
        sPartials1[ GET_SMEM_OFFSET_PARTIALS( tx, ty) ] = partials1[y | tx]; /*All coalesced memory*/
//      sPartials1Tmp[ multBy16(patIdx) | tx ] = partials1[y | tx];
        sPartials2[multBy16(patIdx) | tx] = partials2[y | tx];
    } else {
        sPartials1[ GET_SMEM_OFFSET_PARTIALS( tx, ty) ] = 0;
//        sPartials1Tmp[multBy16(patIdx) | tx] = 0;
        sPartials2[multBy16(patIdx) | tx] = 0;
//        tmpAcc[GET_SMEM_OFFSET_PARTIALS(tx % 4, patIdx * 4 + (tx / 4))] = 0;
    }

    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrix1 = matrices1 + x2; /*Points to *this* matrix*/
    const KW_GLOBAL_VAR REAL* KW_RESTRICT matrix2 = matrices2 + x2;
    KW_LOCAL_MEM REAL sMatrix1[16]; /*Load values into shared memory*/
    KW_LOCAL_MEM REAL sMatrix2[16];
    if (patIdx == 0 ) {
        /* Write transpose of both matrices since M is loaded row-wise */
        sMatrix1[tx] = matrix1[tx];
        sMatrix2[multBy4(state) | pat] = matrix2[tx];
    }
    KW_LOCAL_FENCE;

//    tmpAcc[ tx + patIdx * 16 ] = sPartials1[ tx + patIdx * 16 ];

    double a2 = 0, b2 = 0, res22 = 0, res21 = 0, a1 = 0, b1 = 0, res11 = 0, res12 = 0;

    int reg_row = tx % 4;
    int reg_col = tx / 4;
    int reg_row_partials = tx % 16;
    int reg_col_partials = (patIdx % 2);

    if (patIdx % 2 == 0) {
        a2 = sMatrix2[reg_col * 4 + reg_row];
    } else {
        a2 = 0;
    }

    b2 = sPartials2[warpIdx * 32 + reg_col_partials * 16 + reg_row_partials];

    asm("mma.sync.aligned.m8n8k4.row.col.f64.f64.f64.f64 {%0,%1}, {%2}, {%3}, {%4,%5};\n"
            : "=d"(res21), "=d"(res22)
            : "d"(a2), "d"(b2), "d"(res21), "d"(res22));

    int partials1Index = (patIdx/2) * 32 + ((laneId * 2) % 8) * 4 + (laneId * 2) / WMMA_N;
//    int partials1Tx1 = partials1Index % 16;
//    int partials1Ty1 = partials1Index / 16;
//    int partials1Tx2 = (partials1Index + 4) % 16;
//    int partials1Ty2 = (partials1Index + 4) / 16;
    int partials1State1 = partials1Index % 16;
    int partials1PatIdx1 = partials1Index / 16;
    int partials1State2 = (partials1Index + 4) % 16;
    int partials1PatIdx2 = (partials1Index + 4) / 16;

//        int warpState = tx / warpSize;
//        int warpPattern = patIdx;
//        int warpIdx = warpState + warpPattern * 0.5; // blockDim.x is half a warp
//        int laneId = tx + (patIdx % 2) * 16;
//    int partials1WarpIdx1 = (partials1State1 / warpSize) + partials1PatIdx1 * 0.5;
//    int partials1LaneId1 = partials1State1 + (partials1PatIdx1 % 2) * 16;
//    int partials1WarpIdx2 = (partials1State2 / warpSize) + partials1PatIdx2 * 0.5;
//    int partials1LaneId2 = partials1State2 + (partials1PatIdx2 % 2) * 16;



    KW_LOCAL_FENCE;
    // TODO: Permute ShM to avoid bank conflicts. Existing permute does not work!!
    if(laneId < 16) { // Ignore lower half of matrices. We only need 4 x 8

//        sPartials1Tmp[partials1Index] = sPartials1Tmp[partials1Index] * res21;
//        sPartials1Tmp[partials1Index + 4] = sPartials1Tmp[partials1Index + 4] * res22;

        sPartials1[ GET_SMEM_OFFSET_PARTIALS( partials1State1, partials1PatIdx1 ) ] = sPartials1[ GET_SMEM_OFFSET_PARTIALS( partials1State1, partials1PatIdx1) ] * res21;
        sPartials1[ GET_SMEM_OFFSET_PARTIALS( partials1State2, partials1PatIdx2) ] = sPartials1[ GET_SMEM_OFFSET_PARTIALS( partials1State2, partials1PatIdx2) ] * res22;

    }

    KW_LOCAL_FENCE;

//    tmpAcc[tx + patIdx * 16] = sPartials1[GET_SMEM_OFFSET_PARTIALS( laneId, warpIdx)];

    if (patIdx % 2 == 0) {
        a1 = sMatrix1[reg_col * 4 + reg_row];
    } else {
        a1 = 0;
    }

    partials1Index = warpIdx * 32 + reg_col_partials * 16 + reg_row_partials;
    int partials1State = partials1Index % 16;
    int partials1PatIdx = partials1Index / 16;
//    int partials1WarpIdx = ((partials1State / warpSize) + partials1PatIdx * 0.5);
//    int partials1LaneId = (partials1State + (partials1PatIdx % 2) * 16);

    b1 = sPartials1[GET_SMEM_OFFSET_PARTIALS(partials1State, partials1PatIdx)];

    asm("mma.sync.aligned.m8n8k4.row.col.f64.f64.f64.f64 {%0,%1}, {%2}, {%3}, {%4,%5};\n"
            : "=d"(res11), "=d"(res12)
            : "d"(a1), "d"(b1), "d"(res11), "d"(res12));

    int patternBlock = __umul24(KW_GROUP_ID_0, PATTERN_BLOCK_SIZE * 4);
    u = patternBlock * 4 + deltaPartialsByMatrix;

    if(laneId < 16) {
        if(patternBlock + patIdx * 4 + ((laneId * 2) % 8) < endPattern){
            partials3[u + (patIdx/2) * 32 + ((laneId * 2) % 8) * 4 + (laneId * 2) / WMMA_N] = res11;
        }

        if(patternBlock + patIdx * 4 + ((laneId * 2) % 8) + 1 < endPattern)
            partials3[u + 4 + (patIdx/2) * 32 + ((laneId * 2) % 8) * 4 + (laneId * 2) / WMMA_N] = res12;

    }

#endif // FW_OPENCL_CPU
}